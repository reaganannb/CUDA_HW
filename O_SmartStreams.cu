#include "hip/hip_runtime.h"
// Name: Reagan Burleson
// Two streams overlapped smartly 
// nvcc 15SmartStreams.cu -o temp
/*
 What to do:
 Read about CUDA stream cooperation.

This code provides most of the setup needed to create two CUDA streams. 
Complete the implementation by replacing all the ???s.

Once the two streams are working, overlap them in a smart way to improve performance.
*/

/*
 Purpose:
 To learn how to use CUDA streams intelligently.
*/

// Include files
#include <sys/time.h>
#include <stdio.h>

// Defines
#define DATA_CHUNKS (1024*1024) 
#define ENTIRE_DATA_SET (20*DATA_CHUNKS)
#define MAX_RANDOM_NUMBER 1000
#define BLOCK_SIZE 256

//Globals
dim3 BlockSize; //This variable will hold the Dimensions of your block
dim3 GridSize; //This variable will hold the Dimensions of your grid
float *NumbersOnGPU, *PageableNumbersOnCPU, *PageLockedNumbersOnCPU;
float *A_CPU, *B_CPU, *C_CPU; //CPU pointers
float *A0_GPU, *B0_GPU, *C0_GPU, *A1_GPU, *B1_GPU, *C1_GPU; //GPU pointers
hipEvent_t StartEvent, StopEvent;
hipStream_t Stream0, Stream1;

//Function prototypes
void cudaErrorCheck(const char *, int);
void setUpCudaDevices();
void allocateMemory();
void loadData();
void cleanUp();
__global__ void trigAdditionGPU(float *, float *, float *, int );

// This check to see if an error happened in your CUDA code. It tell you what it thinks went wrong,
// and what file and line it occured on.
void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

//This will be the layout of the parallel space we will be using.
void setUpCudaDevices()
{
	hipEventCreate(&StartEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventCreate(&StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	
	hipDeviceProp_t prop;
	int whichDevice;
	
	hipGetDevice(&whichDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	
	hipGetDeviceProperties(&prop, whichDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	
	if(prop.deviceOverlap != 1)
	{
		printf("\n GPU will not handle overlaps so no speedup from streams");
		printf("\n Good bye.");
		exit(0);
	}
	
	hipStreamCreate(&Stream0);
	cudaErrorCheck(__FILE__, __LINE__);
	hipStreamCreate(&Stream1);
	cudaErrorCheck(__FILE__, __LINE__);
	
	BlockSize.x = BLOCK_SIZE;
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	if(DATA_CHUNKS%BLOCK_SIZE != 0)
	{
		printf("\n Data chunks do not divide evenly by block size, sooo this program will not work.");
		printf("\n Good bye.");
		exit(0);
	}
	GridSize.x = DATA_CHUNKS/BLOCK_SIZE;
	GridSize.y = 1;
	GridSize.z = 1;	
}

//Sets a side memory on the GPU and CPU for our use.
void allocateMemory()
{	
	//Allocate Device (GPU) Memory
	hipMalloc(&A0_GPU,DATA_CHUNKS*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&B0_GPU,DATA_CHUNKS*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&C0_GPU,DATA_CHUNKS*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&A1_GPU,DATA_CHUNKS*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&B1_GPU,DATA_CHUNKS*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&C1_GPU,DATA_CHUNKS*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	
	//Allocate page locked Host (CPU) Memory
	hipHostAlloc(&A_CPU, ENTIRE_DATA_SET*sizeof(float), hipHostMallocDefault);
	cudaErrorCheck(__FILE__, __LINE__);
	hipHostAlloc(&B_CPU, ENTIRE_DATA_SET*sizeof(float), hipHostMallocDefault);
	cudaErrorCheck(__FILE__, __LINE__);
	hipHostAlloc(&C_CPU, ENTIRE_DATA_SET*sizeof(float), hipHostMallocDefault);
	cudaErrorCheck(__FILE__, __LINE__);
}

void loadData()
{
	time_t t;
	srand((unsigned) time(&t));
	
	for(int i = 0; i < ENTIRE_DATA_SET; i++)
	{		
		A_CPU[i] = MAX_RANDOM_NUMBER*rand()/RAND_MAX;
		B_CPU[i] = MAX_RANDOM_NUMBER*rand()/RAND_MAX;	
	}
}

//Cleaning up memory after we are finished.
void cleanUp()
{
	hipFree(A0_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(B0_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(C0_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(A1_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(B1_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(C1_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	
	hipHostFree(A_CPU);
	cudaErrorCheck(__FILE__, __LINE__);
	hipHostFree(B_CPU);
	cudaErrorCheck(__FILE__, __LINE__);
	hipHostFree(C_CPU);
	cudaErrorCheck(__FILE__, __LINE__);
	
	hipEventDestroy(StartEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventDestroy(StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	
	hipStreamDestroy(Stream0);
	cudaErrorCheck(__FILE__, __LINE__);
	hipStreamDestroy(Stream1);
	cudaErrorCheck(__FILE__, __LINE__);
}

__global__ void trigAdditionGPU(float *a, float *b, float *c, int n)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(id < n)
	{
		c[id] = sin(a[id]) + cos(b[id]);
	}
}

int main()
{
	float timeEvent;
	
	setUpCudaDevices();
	allocateMemory();
	loadData();
	
	hipEventRecord(StartEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	
	for(int i = 0; i < ENTIRE_DATA_SET; i += DATA_CHUNKS*2)
	{
		//Offsets
        size_t offset0 = i;
        size_t offset1 = i + DATA_CHUNKS;

        //Handle a non-multiple safely 
        int n0 = (int)min((size_t)DATA_CHUNKS, (size_t)(ENTIRE_DATA_SET - offset0));
        int n1 = (int)min((size_t)DATA_CHUNKS, (size_t)(ENTIRE_DATA_SET - offset1));

        //Stream 0
        hipMemcpyAsync(A0_GPU, A_CPU + offset0, n0 * sizeof(float), hipMemcpyHostToDevice, Stream0);
        cudaErrorCheck(__FILE__, __LINE__);
        hipMemcpyAsync(B0_GPU, B_CPU + offset0, n0 * sizeof(float), hipMemcpyHostToDevice, Stream0);
        cudaErrorCheck(__FILE__, __LINE__);

        trigAdditionGPU<<<GridSize, BlockSize, 0, Stream0>>>(A0_GPU, B0_GPU, C0_GPU, n0);
        cudaErrorCheck(__FILE__, __LINE__);

        hipMemcpyAsync(C_CPU + offset0, C0_GPU, n0 * sizeof(float), hipMemcpyDeviceToHost, Stream0);
        cudaErrorCheck(__FILE__, __LINE__);

        //Stream 1
        if (offset1 < ENTIRE_DATA_SET)
        {
            hipMemcpyAsync(A1_GPU, A_CPU + offset1, n1 * sizeof(float), hipMemcpyHostToDevice, Stream1);
            cudaErrorCheck(__FILE__, __LINE__);
            hipMemcpyAsync(B1_GPU, B_CPU + offset1, n1 * sizeof(float), hipMemcpyHostToDevice, Stream1);
            cudaErrorCheck(__FILE__, __LINE__);

            trigAdditionGPU<<<GridSize, BlockSize, 0, Stream1>>>(A1_GPU, B1_GPU, C1_GPU, n1);
            cudaErrorCheck(__FILE__, __LINE__);

            hipMemcpyAsync(C_CPU + offset1, C1_GPU, n1 * sizeof(float), hipMemcpyDeviceToHost, Stream1);
            cudaErrorCheck(__FILE__, __LINE__);
        }

	}
	
	// Make the CPU wait until the Streams have finishd before it continues.
	hipStreamSynchronize(Stream0);
	hipStreamSynchronize(Stream1);
	
	hipEventRecord(StopEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	// Make the CPU wiat until this event finishes so the timing will be correct.
	hipEventSynchronize(StopEvent); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventElapsedTime(&timeEvent, StartEvent, StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	printf("\n Time on GPU = %3.1f milliseconds", timeEvent);
	
	
	printf("\n");
	//You're done so cleanup your mess.
	cleanUp();	
	
	return(0);
}
